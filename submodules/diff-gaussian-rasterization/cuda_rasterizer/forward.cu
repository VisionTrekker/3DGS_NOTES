#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
    // The implementation is loosely based on code for 
    // "Differentiable Point-Based Radiance Fields for 
    // Efficient View Synthesis" by Zhang et al. (2022)
    glm::vec3 pos = means[idx];
    glm::vec3 dir = pos - campos;
    dir = dir / glm::length(dir);

    glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
    glm::vec3 result = SH_C0 * sh[0];

    if (deg > 0)
    {
        float x = dir.x;
        float y = dir.y;
        float z = dir.z;
        result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

        if (deg > 1)
        {
            float xx = x * x, yy = y * y, zz = z * z;
            float xy = x * y, yz = y * z, xz = x * z;
            result = result +
                SH_C2[0] * xy * sh[4] +
                SH_C2[1] * yz * sh[5] +
                SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
                SH_C2[3] * xz * sh[7] +
                SH_C2[4] * (xx - yy) * sh[8];

            if (deg > 2)
            {
                result = result +
                    SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
                    SH_C3[1] * xy * z * sh[10] +
                    SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
                    SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
                    SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
                    SH_C3[5] * z * (xx - yy) * sh[14] +
                    SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
            }
        }
    }
    result += 0.5f;

    // RGB colors are clamped to positive values. If values are
    // clamped, we need to keep track of this for the backward pass.
    clamped[3 * idx + 0] = (result.x < 0);
    clamped[3 * idx + 1] = (result.y < 0);
    clamped[3 * idx + 2] = (result.z < 0);
    return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
    // The following models the steps outlined by equations 29
    // and 31 in "EWA Splatting" (Zwicker et al., 2002). 
    // Additionally considers aspect / scaling of viewport.
    // Transposes used to account for row-/column-major conventions.
    float3 t = transformPoint4x3(mean, viewmatrix);

    const float limx = 1.3f * tan_fovx;
    const float limy = 1.3f * tan_fovy;
    const float txtz = t.x / t.z;
    const float tytz = t.y / t.z;
    t.x = min(limx, max(-limx, txtz)) * t.z;
    t.y = min(limy, max(-limy, tytz)) * t.z;

    glm::mat3 J = glm::mat3( // 射影变换的仿射近似的雅可比矩阵
        focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
        0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
        0, 0, 0);

    glm::mat3 W = glm::mat3(
        viewmatrix[0], viewmatrix[4], viewmatrix[8],
        viewmatrix[1], viewmatrix[5], viewmatrix[9],
        viewmatrix[2], viewmatrix[6], viewmatrix[10]);

    glm::mat3 T = W * J;

    glm::mat3 Vrk = glm::mat3(
        cov3D[0], cov3D[1], cov3D[2],
        cov3D[1], cov3D[3], cov3D[4],
        cov3D[2], cov3D[4], cov3D[5]);

    glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

    // Apply low-pass filter: every Gaussian should be at least
    // one pixel wide/high. Discard 3rd row and column. 直接丢掉第三行第三列
    cov[0][0] += 0.3f;
    cov[1][1] += 0.3f;
    return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) }; // 对称矩阵，只返回 3 个足矣
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
    // Create scaling matrix
    glm::mat3 S = glm::mat3(1.0f);
    S[0][0] = mod * scale.x;
    S[1][1] = mod * scale.y;
    S[2][2] = mod * scale.z;

    // Normalize quaternion to get valid rotation
    glm::vec4 q = rot;// / glm::length(rot);
    float r = q.x;
    float x = q.y;
    float y = q.z;
    float z = q.w;

    // Compute rotation matrix from quaternion
    glm::mat3 R = glm::mat3(
        1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
        2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
        2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
    );

    glm::mat3 M = S * R;

    // Compute 3D world covariance matrix Sigma
    glm::mat3 Sigma = glm::transpose(M) * M;

    // Covariance is symmetric, only store upper right
    cov3D[0] = Sigma[0][0];
    cov3D[1] = Sigma[0][1];
    cov3D[2] = Sigma[0][2];
    cov3D[3] = Sigma[1][1];
    cov3D[4] = Sigma[1][2];
    cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
    const float* orig_points,
    const glm::vec3* scales,
    const float scale_modifier,
    const glm::vec4* rotations,
    const float* opacities,
    const float* shs,
    bool* clamped,
    const float* cov3D_precomp,
    const float* colors_precomp,
    const float* viewmatrix,
    const float* projmatrix,
    const glm::vec3* cam_pos,
    const int W, int H,
    const float tan_fovx, float tan_fovy,
    const float focal_x, float focal_y,
    int* radii,
    float2* points_xy_image,
    float* depths,
    float* cov3Ds,
    float* rgb,
    float4* conic_opacity,
    const dim3 grid,
    uint32_t* tiles_touched,
    bool prefiltered)
{
    // 在 kernel function 中，代码在 thread 级别上运行，通过下面的语句确定自己处理的是哪个高斯（注意：后面的代码都是针对单个高斯了）
    auto idx = cg::this_grid().thread_rank(); 
    if (idx >= P)
        return;

    // Initialize radius and touched tiles to 0. If this isn't changed,
    // this Gaussian will not be processed further.
    radii[idx] = 0;
    tiles_touched[idx] = 0;

    // Perform near culling, quit if outside. 给定指定的相机姿势，此步骤确定哪些3D高斯位于相机的视锥体之外。这样做可以确保在后续计算中不涉及给定视图之外的3D高斯，从而节省计算资源
    float3 p_view;
    if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
        return;

    // Transform point by projecting  以下代码将3D高斯（椭球）被投影到2D图像空间（椭圆），存储必要的变量供后续渲染使用
    // 首先投影 xyz（均值）
    float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] }; // xyz
    float4 p_hom = transformPoint4x4(p_orig, projmatrix);
    float p_w = 1.0f / (p_hom.w + 0.0000001f);
    float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w }; // 齐次坐标系的归一化，除法都把它实现为乘法以加速

    // 然后投影协方差矩阵，先要 computeCov3D 把它从四元数和缩放因子还原出来，然后再 computeCov3D
    // If 3D covariance matrix is precomputed, use it, otherwise compute
    // from scaling and rotation parameters. 计算世界坐标系下的协方差矩阵
    const float* cov3D;
    if (cov3D_precomp != nullptr)
    {
        cov3D = cov3D_precomp + idx * 6;
    }
    else
    {
        computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
        cov3D = cov3Ds + idx * 6;
    }

    // Compute 2D screen-space covariance matrix 计算视图变换和投影变换后的协方差矩阵
    float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

    // Invert covariance (EWA algorithm)
    float det = (cov.x * cov.z - cov.y * cov.y); // 行列式
    if (det == 0.0f) // 非满秩，直接返回
        return;
    float det_inv = 1.f / det;
    float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv }; // 2D 协方差矩阵的逆

    // Compute extent in screen space (by finding eigenvalues of
    // 2D covariance matrix). Use extent to compute a bounding rectangle
    // of screen-space tiles that this Gaussian overlaps with. Quit if
    // rectangle covers 0 tiles. 
    float mid = 0.5f * (cov.x + cov.z);
    float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
    float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));  // 这里简单推导一下就知道它在算什么了。矩阵的特征值代表了椭圆的两个半轴（回忆 PCA）
    float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));  // 用圆来近似椭圆，稍微往大一点算（但 3 倍不会太大吗？还是我理解错了？）
    float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) }; // 转换到图片坐标系，uv 坐标
    uint2 rect_min, rect_max;
    getRect(point_image, my_radius, rect_min, rect_max, grid); // 求交，rect_minmax 代表在 tile_grid 中的坐标
    if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
        return;

    // If colors have been precomputed, use them, otherwise convert
    // spherical harmonics coefficients to RGB color.
    if (colors_precomp == nullptr)
    {
        glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
        rgb[idx * C + 0] = result.x;
        rgb[idx * C + 1] = result.y;
        rgb[idx * C + 2] = result.z;
    }

    // Store some useful helper data for the next steps.
    depths[idx] = p_view.z; // 用于排序的深度就是投影后在 [-1,1]^3 投影坐标系下的深度
    radii[idx] = my_radius;
    points_xy_image[idx] = point_image; // uv 坐标
    // Inverse 2D covariance and opacity neatly pack into one float4
    conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };  // 前三个将被用于计算高斯的指数部分从而得到 prob（查询点到该高斯的距离->prob，例如，若查询点位于该高斯的中心则 prob 为 1）。最后一个是该高斯本身的密度。
    tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
    const uint2* __restrict__ ranges,
    const uint32_t* __restrict__ point_list,
    int W, int H,
    const float2* __restrict__ points_xy_image,
    const float* __restrict__ features,
    const float4* __restrict__ conic_opacity,
    float* __restrict__ final_T,
    uint32_t* __restrict__ n_contrib,
    const float* __restrict__ bg_color,
    float* __restrict__ out_color)
{
    // 现在是 thread 级别，在一个 tile 内，每个 thread 与一个 Gaussion 计算
    // Identify current tile and associated min/max pixel range.
    auto block = cg::this_thread_block();
    uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
    uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
    uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
    uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
    uint32_t pix_id = W * pix.y + pix.x;
    float2 pixf = { (float)pix.x, (float)pix.y };

    // Check if this thread is associated with a valid pixel or outside.
    bool inside = pix.x < W && pix.y < H;
    // Done threads can help with fetching, but don't rasterize
    bool done = !inside;

    // 下面要把 range 内的 Gaussion 取出来放到 shared memory（每个线程并行读取）
    // 一共有 toDo 这么多个 Gaussions 需要处理，然后一共有 BLOCK_SIZE 那么多个 threads，每个取一个的话就只用 rounds 轮
    // 另外它不是把所有全部搬完了再处理，而是一轮搬完就处理一轮，下一轮搬的时候覆盖掉上一轮，对空间友好（shared memory 不用开太大）
    // Load start/end range of IDs to process in bit sorted list.
    uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
    const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
    int toDo = range.y - range.x; 

    // Allocate storage for batches of collectively fetched data.
    __shared__ int collected_id[BLOCK_SIZE];
    __shared__ float2 collected_xy[BLOCK_SIZE];
    __shared__ float4 collected_conic_opacity[BLOCK_SIZE];

    // Initialize helper variables
    float T = 1.0f;
    uint32_t contributor = 0;
    uint32_t last_contributor = 0;
    float C[CHANNELS] = { 0 };

    // Iterate over batches until all done or range is complete
    for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
    {
        // End if entire block votes that it is done rasterizing
        int num_done = __syncthreads_count(done);
        if (num_done == BLOCK_SIZE)
            break;

        // Collectively fetch per-Gaussian data from global to shared
        int progress = i * BLOCK_SIZE + block.thread_rank(); // i * BLOCK_SIZE: 之前轮已经搬完了的位置；block.thread_rank(): 这一轮我这个 thread 的序号
        if (range.x + progress < range.y)
        {
            int coll_id = point_list[range.x + progress];
            collected_id[block.thread_rank()] = coll_id;                           // Gaussion id (uint32_t)
            collected_xy[block.thread_rank()] = points_xy_image[coll_id];          // Gaussion 在图片坐标系下的坐标 (float2)
            collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id]; // Gaussion 2D 协方差矩阵的逆和不透明度
        }
        block.sync(); // 等待所有线程都搬完，下面开始处理

        // Iterate over current batch
        for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
        {
            // Keep track of current position in range
            contributor++;

            // Resample using conic matrix (cf. "Surface 
            // Splatting" by Zwicker et al., 2001)
            float2 xy = collected_xy[j];
            float2 d = { xy.x - pixf.x, xy.y - pixf.y };
            float4 con_o = collected_conic_opacity[j];
            float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y; // 指数上方的结果
            if (power > 0.0f) // 高斯分布大于 1 了（像素点出现在这个高斯的几率大于 1，说明这个高斯不正常）
                continue;

            // Eq. (2) from 3D Gaussian splatting paper.
            // Obtain alpha by multiplying with Gaussian opacity
            // and its exponential falloff from mean.
            // Avoid numerical instabilities (see paper appendix). 
            float alpha = min(0.99f, con_o.w * exp(power));  // opacity * 像素点出现在这个高斯的几率
            if (alpha < 1.0f / 255.0f)  // 太小了就当成透明的
                continue;
            float test_T = T * (1 - alpha);  // alpha合成的系数
            if (test_T < 0.0001f)  // 累乘不透明度到一定的值，标记这个像素的渲染结束
            {
                done = true;
                continue;
            }

            // Eq. (3) from 3D Gaussian splatting paper.
            for (int ch = 0; ch < CHANNELS; ch++)
                C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T; // C = ∑_{i ∈ N} c_i α'_i ∏_{j=1}^{i-1} (1 - α'_j)

            T = test_T;

            // Keep track of last range entry to update this
            // pixel.
            last_contributor = contributor;
        }
    }

    // All threads that treat valid pixel write out their final
    // rendering data to the frame and auxiliary buffers.
    if (inside)
    {
        final_T[pix_id] = T;                   // 用于反向传播计算梯度，储存在 imgState 中返回
        n_contrib[pix_id] = last_contributor;  // 记录数量，用于提前停止计算
        for (int ch = 0; ch < CHANNELS; ch++)
            out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch]; // 再加上背景颜色
    }
}

void FORWARD::render(
    const dim3 grid, dim3 block,
    const uint2* ranges,
    const uint32_t* point_list,
    int W, int H,
    const float2* means2D,
    const float* colors,
    const float4* conic_opacity,
    float* final_T,
    uint32_t* n_contrib,
    const float* bg_color,
    float* out_color)
{
    renderCUDA<NUM_CHANNELS> << <grid, block>> > (
        ranges,
        point_list,
        W, H,
        means2D,
        colors,
        conic_opacity,
        final_T,
        n_contrib,
        bg_color,
        out_color);
}

void FORWARD::preprocess(int P, int D, int M,
    const float* means3D,
    const glm::vec3* scales,
    const float scale_modifier,
    const glm::vec4* rotations,
    const float* opacities,
    const float* shs,
    bool* clamped,
    const float* cov3D_precomp,
    const float* colors_precomp,
    const float* viewmatrix,
    const float* projmatrix,
    const glm::vec3* cam_pos,
    const int W, int H,
    const float focal_x, float focal_y,
    const float tan_fovx, float tan_fovy,
    int* radii,
    float2* means2D,
    float* depths,
    float* cov3Ds,
    float* rgb,
    float4* conic_opacity,
    const dim3 grid,
    uint32_t* tiles_touched,
    bool prefiltered)
{
    // 封装，调用 kernel function，一个 block 算 256 个高斯，其中的每个高斯对应一个 thread 来处理
    preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256>> > (
        P, D, M,
        means3D,
        scales,
        scale_modifier,
        rotations,
        opacities,
        shs,
        clamped,
        cov3D_precomp,
        colors_precomp,
        viewmatrix, 
        projmatrix,
        cam_pos,
        W, H,
        tan_fovx, tan_fovy,
        focal_x, focal_y,
        radii,
        means2D,
        depths,
        cov3Ds,
        rgb,
        conic_opacity,
        grid,
        tiles_touched,
        prefiltered
        );
}